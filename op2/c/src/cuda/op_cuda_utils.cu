#include "hip/hip_runtime.h"
#include "op_lib_core.h"
#include "op_lib_cpp.h"
#include "op_cuda_rt_support.h"
#include "op_cuda_utils.h"

__device__ int pos(int row, int col, int* rowptr, int* colidx)
{
  for ( int k = rowptr[row]; k < rowptr[row+1]; k++ )
    if ( colidx[k] == col )
      return k;
  return INT_MAX;
}

template<class T>
__global__ void op_lma_to_csr_dev (T *lma, T *data,
                                   int *rowptr,
                                   int *colidx,
                                   int *rmap,
                                   int rmapdim,
                                   int *cmap,
                                   int cmapdim,
                                   int nelems)
{
  if ( threadIdx.x + blockIdx.x * blockDim.x > nelems * rmapdim * cmapdim )
    return;

  int n;
  int e;
  int i;
  int j;

  int entry_per_ele = rmapdim * cmapdim;

  int offset;
  int row;
  int col;
  for ( n = threadIdx.x; n < nelems * entry_per_ele; n += blockDim.x )
  {
    e = n / entry_per_ele;
    i = (n - e * entry_per_ele) / rmapdim;
    j = (n - e * entry_per_ele - i * cmapdim);

    row = rmap[e * rmapdim + i];
    col = cmap[e * cmapdim + j];

    offset = pos(row, col, rowptr, colidx);
    op_atomic_add(data + offset, lma[n]);
  }
}

template<class T>
__host__ void op_mat_lma_to_csr(op_arg arg, op_set set)
{
  op_mat mat = arg.mat;
  op_sparsity sparsity = mat->sparsity;
  op_map rmap = sparsity->rowmap;
  int rmapdim = rmap->dim;
  op_map cmap = sparsity->colmap;
  int cmapdim = cmap->dim;
  int * rowptr = sparsity->rowptr;
  int * colidx = sparsity->colidx;
  int nelems = set->size;

  if ( rmap->map_d == NULL ) {
    op_cpHostToDevice ((void **)&(rmap->map_d),
        (void **)&(rmap->map),
        sizeof(int) * rmapdim * nelems);
  }

  if ( cmap->map_d == NULL ) {
    op_cpHostToDevice((void **)&(cmap->map_d),
        (void **)&(cmap->map),
        sizeof(int) * cmapdim * nelems);
  }
  op_lma_to_csr_dev<<<128, 128>>> (((T *)mat->lma_data),
                                   ((T *)mat->data),
                                   rowptr,
                                   colidx,
                                   rmap->map_d,
                                   rmapdim,
                                   cmap->map_d,
                                   cmapdim,
                                   nelems);
}

__host__ void op_mat_lma_to_csr(float *dummy, op_arg arg, op_set set)
{
  op_mat_lma_to_csr<float>(arg, set);
}

__host__ void op_mat_lma_to_csr(double *dummy, op_arg arg, op_set set)
{
  op_mat_lma_to_csr<double>(arg, set);
}
