#include "hip/hip_runtime.h"
#include "op_lib_core.h"
#include "op_lib_mat.h"
#include "op_lib_cpp.h"
#include "op_cuda_rt_support.h"
#include "op_cuda_utils.h"

__device__ int pos(int row, int col, int* rowptr, int* colidx)
{
  for ( int k = rowptr[row]; k < rowptr[row+1]; k++ )
    if ( colidx[k] == col )
      return k;
  return INT_MAX;
}

template<class T>
__global__ void op_lma2csr_dev(T * lma, T * data,
                               int * lmaidx,
                               int * csr2lma,
                               int nentries)
{

    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if ( id >= nentries ) return;

    for ( int i = lmaidx[id]; i < lmaidx[id+1]; i++ )
        data[id] += lma[csr2lma[i]];
}

template<class T>
__host__ void op_mat_lma2csr (op_arg arg)
{
    op_mat mat = arg.mat;
    op_sparsity sparsity = mat->sparsity;

    int nthread = 128;
    int nentries = sparsity->total_nz;

    int nblock = nentries / nthread + 1;

    op_lma2csr_dev<<<nblock,nthread>>>((T *)mat->lma_data,
                                       (T *)mat->data,
                                       sparsity->lmaidx,
                                       sparsity->csr2lma,
                                       nentries);
}

__host__ void op_mat_lma2csr(float *dummy, op_arg arg)
{
    op_mat_lma2csr<float>(arg);
}

__host__ void op_mat_lma2csr(double *dummy, op_arg arg)
{
    op_mat_lma2csr<double>(arg);
}

template<class T>
__global__ void op_lma_to_csr_dev (T *lma, T *data,
                                   int *rowptr,
                                   int *colidx,
                                   int *rmap,
                                   int rmapdim,
                                   int *cmap,
                                   int cmapdim,
                                   int nelems)
{
  int n;
  int e;
  int i;
  int j;
  int entry_per_ele;
  int offset;
  int row;
  int col;
  entry_per_ele = rmapdim * cmapdim;

  n = threadIdx.x + blockIdx.x * blockDim.x;
  if ( n >= nelems * entry_per_ele ) return;

  e = n / entry_per_ele;
  i = (n - e * entry_per_ele) / rmapdim;
  j = (n - e * entry_per_ele - i * cmapdim);

  row = rmap[e * rmapdim + i];
  col = cmap[e * cmapdim + j];

  offset = pos(row, col, rowptr, colidx);
  op_atomic_add(data + offset, lma[n]);
}

template<class T>
__host__ void op_mat_lma_to_csr(op_arg arg, op_set set)
{
  op_mat mat = arg.mat;
  op_sparsity sparsity = mat->sparsity;
  op_map rmap = sparsity->rowmap;
  int rmapdim = rmap->dim;
  op_map cmap = sparsity->colmap;
  int cmapdim = cmap->dim;
  int * rowptr = sparsity->rowptr;
  int * colidx = sparsity->colidx;
  int nelems = set->size;

  if ( rmap->map_d == NULL ) {
    op_cpHostToDevice ((void **)&(rmap->map_d),
        (void **)&(rmap->map),
        sizeof(int) * rmapdim * nelems);
  }

  if ( cmap->map_d == NULL ) {
    op_cpHostToDevice((void **)&(cmap->map_d),
        (void **)&(cmap->map),
        sizeof(int) * cmapdim * nelems);
  }
  int nthread = 128;
  int nblock = (nelems * rmapdim * cmapdim) / nthread + 1;
  op_lma_to_csr_dev<<<nblock, nthread>>> (((T *)mat->lma_data),
                                          ((T *)mat->data),
                                          rowptr,
                                          colidx,
                                          rmap->map_d,
                                          rmapdim,
                                          cmap->map_d,
                                          cmapdim,
                                          nelems);
}

__host__ void op_mat_lma_to_csr(float *dummy, op_arg arg, op_set set)
{
  op_mat_lma_to_csr<float>(arg, set);
}

__host__ void op_mat_lma_to_csr(double *dummy, op_arg arg, op_set set)
{
  op_mat_lma_to_csr<double>(arg, set);
}
