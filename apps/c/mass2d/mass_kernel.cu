#include "hip/hip_runtime.h"
#include "op_lib_cpp.h"
#include "op_cuda_rt_support.h"
__device__
#include "mass_cuda.h"

__global__ void op_cuda_mass(float *mat_data,
                             int *rowptr,
                             int *colptr,
                             int nrow,
                             int *map,
                             int map_dim,
                             int *nelems,
                             int *blockoffset,
                             int nblocks,
                             float *data,
                             int dim)
{
  extern __shared__ char shared[];

  __shared__ float *data_s;
  __shared__ int *map_s;
  if ( blockIdx.x >= nblocks ) return;

  int nelem = nelems[blockIdx.x];
  int boffset = blockoffset[blockIdx.x];

  if ( threadIdx.x == 0 ) {
    data_s = (float *)&shared[0];
    map_s = (int *)&shared[ROUND_UP(nelem * map_dim * dim * sizeof(float))];
  }

  __syncthreads();
  // Needs nelem * map_dim * dim floats for data.
  // Plus a further nelem * map_dim ints for the map data
  // So for P1 triangles this requires
  // 6 * nelem * sizeof(float) + 3 * nelem * sizeof(int) bytes (plus
  // slop for 16-byte alignment)
  for ( int n = threadIdx.x; n < nelem; n+= blockDim.x ) {
    for ( int k = 0; k < map_dim; k++ ) {
      data_s[dim*(n * map_dim + k)] = data[dim*map[(n+boffset)*map_dim + k]];
      data_s[dim*(n * map_dim + k) + 1] = data[dim*map[(n+boffset)*map_dim + k] + 1];
      map_s[n * map_dim + k] = map[(n + boffset)*map_dim + k];
    }
  }
  __syncthreads();
  float entry;
  for ( int k = threadIdx.x; k < nelem * 3 * 3 * 3; k+=blockDim.x ) {
    // k == q + 3*j + 9*i + 27*n
    int n = k / 27;
    int i = (k - 27*n) / 9;
    int j = (k - 27*n - 9*i) / 3;
    int q = k - 27*n - 9*i - 3*j;
    int mapi = map_s[n*map_dim + i];
    int mapj = map_s[n*map_dim + j];
    entry = 0.0f;
    // Compute a single matrix entry
    // Probably best if q is iterated over in the kernel, but iterate
    // over it in the parallel loop as an example.
    mass(&entry, (float (*)[2])(data_s + n*map_dim*dim), i, j, q);
    // Insert matrix entry into global matrix.
    // find column offset
    int offset;
    for (int p = rowptr[mapi]; p < rowptr[mapi+1]; p++ ) {
      if ( colptr[p] == mapj )
        offset = p;
    }
    // To avoid these atomics we'd have to do colour-order traversal
    // of the elements.  And, if q is iterated over here, warp-level
    // reductions.
    atomicAdd(mat_data + offset, entry);
  }
}

void op_par_loop_mass(const char *name, op_set elements, op_sparsity sparsity,
                     op_arg arg_dat)
{
  int *map_d;
  cutilSafeCall(hipMalloc((void **)&map_d,
                           sizeof(int) * arg_dat.map->dim * elements->size));
  cutilSafeCall(hipMemcpy(map_d, arg_dat.map->map,
                           sizeof(int) * arg_dat.map->dim * elements->size,
                           hipMemcpyHostToDevice));

  int nthread = 128;
  int nblocks = 128;
  int nblock = 128;
  int nelems_h[nblock];
  int *nelems_d;
  cutilSafeCall(hipMalloc((void **)&nelems_d,
         nblock * sizeof(int)));

  for ( int i = 0; i < nblock; i++ ) {
      nelems_h[i] = elements->size / nblock;
  }
  // Fix up leftovers
  for ( int i = 0; i < elements->size - nblock * (elements->size/nblock); i++ ) {
      nelems_h[i]++;
  }

  int boffset_h[nblock];
  int *boffset_d;

  boffset_h[0] = 0;
  for ( int i = 1; i < nblock; i++ ) {
      boffset_h[i] = boffset_h[i-1] + nelems_h[i-1];
  }
  cutilSafeCall(hipMemcpy(nelems_d, nelems_h, nblock * sizeof(int),
         hipMemcpyHostToDevice));

  cutilSafeCall(hipMalloc((void **)&boffset_d, nblock * sizeof(int)));
  cutilSafeCall(hipMemcpy(boffset_d, boffset_h, nblock * sizeof(int),
         hipMemcpyHostToDevice));

  int nshared;

  // This all needs to be wrapped in cuda versions of op_decl_sparsity
  // and op_decl_mat
  int *rowptr_d;
  int nrow = sparsity->nrows;
  int nnz = sparsity->rowptr[nrow];
  cutilSafeCall(hipMalloc((void **)&rowptr_d, (nrow+1) * sizeof(int)));
  cutilSafeCall(hipMemcpy(rowptr_d, sparsity->rowptr, (nrow+1) * sizeof(int),
                           hipMemcpyHostToDevice));

  int *colptr_d;
  cutilSafeCall(hipMalloc((void **)&colptr_d, nnz * sizeof(int)));
  cutilSafeCall(hipMemcpy(colptr_d, sparsity->colidx, nnz * sizeof(int),
                           hipMemcpyHostToDevice));
  float *data_d;
  cutilSafeCall(hipMalloc((void **)&data_d, nnz * sizeof(float)));
  cutilSafeCall(hipMemset(data_d, 0, nnz * sizeof(float)));

  nshared = nelems_h[0] * arg_dat.map->dim * arg_dat.dat->dim * sizeof(float)
    + nelems_h[0] * arg_dat.map->dim * sizeof(int);


  op_cuda_mass<<<nblocks, nthread, nshared>>>(data_d,
                                              rowptr_d,
                                              colptr_d,
                                              nrow,
                                              map_d,
                                              arg_dat.map->dim,
                                              nelems_d,
                                              boffset_d,
                                              nblock,
                                              (float *)arg_dat.data_d,
                                              arg_dat.dat->dim);

  // Print out resulting matrix if it comes from 2-element problem
  if ( elements->size == 2 ) {
      float *mat_h = (float *)malloc(nnz * sizeof(float));
      cutilSafeCall(hipMemcpy(mat_h, data_d, nnz * sizeof(float),
                               hipMemcpyDeviceToHost));
      for ( int i = 0; i < nrow; i++ ) {
          printf("row %d: ", i);
          for ( int j = sparsity->rowptr[i]; j < sparsity->rowptr[i+1]; j++ ) {
              printf("(%d, %g) ", sparsity->colidx[j], mat_h[j]);
          }
          printf("\n");
      }
      free(mat_h);
  }
  cutilSafeCall(hipFree(data_d));
  cutilSafeCall(hipFree(rowptr_d));
  cutilSafeCall(hipFree(colptr_d));
  cutilSafeCall(hipFree(map_d));
  cutilSafeCall(hipFree(boffset_d));
  cutilSafeCall(hipFree(nelems_d));
}
